#include "hip/hip_runtime.h"
#include "anime4k_cuda.h"
#include "instrument.h"

#include <stdio.h>
#include <stdlib.h>
#include <strings.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define REGIONW 28
#define REGIONH 28
#define PADDING 2
#define THREADW (REGIONW+2*PADDING)
#define THREADH (REGIONH+2*PADDING)

__constant__ Param cudaParam;
unsigned char *cudaImage;
unsigned char *cudaResult;
__global__ void kernel(unsigned char *src, unsigned char *dst);

Anime4kCuda::Anime4kCuda(
    unsigned int width, unsigned int height, unsigned char *image,
    unsigned int new_width, unsigned int new_height)
{
    param.src_width = width;
    param.src_height = height;
    param.dst_width = new_width;
    param.dst_height = new_height;
    param.src_bytes = 4*width*height*sizeof(unsigned char);
    param.dst_bytes = 4*new_width*new_height*sizeof(unsigned char);
    param.strength_preprocessing = min((float)new_width / width / 6.0f, 1.0f);
    param.strength_push = min((float)new_width / width / 2.0f, 1.0f);
    image_ = image;
    result_ = new unsigned char[param.dst_bytes];

    hipMalloc(&cudaImage, param.src_bytes);
    hipMalloc(&cudaResult, param.dst_bytes);
    hipMemcpyToSymbol(HIP_SYMBOL(cudaParam), &param, sizeof(Param));
}

void Anime4kCuda::run()
{
    dim3 gridDim((param.dst_width+REGIONW-1)/REGIONW, 
                (param.dst_height+REGIONH-1)/REGIONH);
    dim3 blockDim(THREADW,THREADH);
    hipMemcpy(cudaImage, image_, param.src_bytes,hipMemcpyHostToDevice);
    kernel<<<gridDim,blockDim>>>(cudaImage,cudaResult);
    hipMemcpy(result_, cudaResult, param.dst_bytes,hipMemcpyDeviceToHost);
}

Anime4kCuda::~Anime4kCuda()
{
    delete [] result_;
    hipFree(cudaImage);
    hipFree(cudaResult);
}

/**  cuda code **/

__device__ __inline__ float 
interpolate(unsigned char tl, unsigned char tr,
    unsigned char bl, unsigned char br, float f, float g)
{
    float minusf = 1.0f-f;
    float minusg = 1.0f-g;
    float t = ((float)tl*minusf+(float)tr*f)/255.0f;
    float b = ((float)bl*minusf+(float)br*f)/255.0f;
    return t*minusg+b*g;
}

__device__ __inline__ void
enlarge(unsigned char *image, float *enlarged, bool qualified)
{
    int threadId = threadIdx.y*blockDim.x+threadIdx.x;
    int pixelX = blockIdx.x*REGIONW+threadIdx.x-PADDING;
    int pixelY = blockIdx.y*REGIONH+threadIdx.y-PADDING;
    int src_width = cudaParam.src_width;
    int src_height = cudaParam.src_height;
    int dst_width = cudaParam.dst_width;
    int dst_height = cudaParam.dst_height;

    
    if (qualified) {
        // map padded pixel to valid pixel
        if (pixelX<0) pixelX = 0;
        if (pixelX>=dst_width) pixelX=dst_width-1;
        if (pixelY<0) pixelY = 0;
        if (pixelY>=dst_height) pixelY=dst_height-1;
    
        float x = (float) (pixelX * src_width) / (float) dst_width;
        float y = (float) (pixelY * src_height) / (float) dst_height;
        int tlx = floor(x);
        int tly = floor(y);
        float f = x - tlx;
        float g = y - tly;
    
        int tl = 4 * (tly*src_width+tlx);
        int tr = tl + 4;
        int bl = tl + 4*src_width;
        int br = bl + 4;
        
        enlarged[3*threadId] = interpolate(image[tl],image[tr],image[bl],image[br],f,g);
        enlarged[3*threadId+1] = interpolate(image[tl+1],image[tr+1],image[bl+1],image[br+1],f,g);
        enlarged[3*threadId+2] = interpolate(image[tl+2],image[tr+2],image[bl+2],image[br+2],f,g);
    }
}


__device__ __inline__ void
compute_luminance(float *image, float *luminace, bool qualified) 
{
    if (qualified) {
        int threadId = threadIdx.y*blockDim.x+threadIdx.x;
        luminace[threadId] = (image[3*threadId]*2+image[3*threadId+1]*3 
                                + image[3*threadId]) / 6.0f;
    }
}

__device__ __inline__ float 
min3v(float a, float b, float c) {
    return min(a,min(b,c));
}

__device__ __inline__ float 
max3v(float a, float b, float c) {
    return max(a,max(b,c));
}

__device__ __inline__ void 
get_largest(float strength, float *image, float *lum,
    float color[4], int cc, int a, int b, int c)
{
    float minusStrength = 1 - strength;
    float new_lum = lum[cc] * minusStrength +
        ((lum[a] + lum[b] + lum[c]) / 3) * strength;
    
    if (new_lum > color[3]) {
        color[0] = image[cc * 3] * minusStrength +
            ((image[a * 3] + image[b * 3] + image[c * 3]) / 3) * strength;
        color[1] = image[cc * 3 + 1] * minusStrength +
            ((image[a * 3 + 1] + image[b * 3 + 1] + image[c * 3 + 1]) / 3) * strength;
        color[2] = image[cc * 3 + 2] * minusStrength +
            ((image[a * 3 + 2] + image[b * 3 + 2] + image[c * 3 + 2]) / 3) * strength;
        color[3] = new_lum;
    }
}



__device__ __inline__ void
preprocess(float *image, float *lum, float *preprocessed, bool qualified)
{
    int threadId = threadIdx.y*blockDim.x+threadIdx.x;

    float color[4];

    if (qualified) {
        color[0] = image[3 * threadId];
        color[1] = image[3 * threadId + 1];
        color[2] = image[3 * threadId + 2];
        color[3] = lum[threadId];
        
        int cc_ix = threadId;
        int r_ix = cc_ix + 1;
        int l_ix = cc_ix - 1;
        int t_ix = cc_ix - blockDim.x;
        int tl_ix = t_ix - 1;
        int tr_ix = t_ix + 1;
        int b_ix = cc_ix + blockDim.x;
        int bl_ix = b_ix - 1;
        int br_ix = b_ix + 1;

        float strength = cudaParam.strength_preprocessing;
        float cc = lum[cc_ix];
        float r = lum[r_ix];
        float l = lum[l_ix];
        float t = lum[t_ix];
        float tl = lum[tl_ix];
        float tr = lum[tr_ix];
        float b = lum[b_ix];
        float bl = lum[bl_ix];
        float br = lum[br_ix];

        float max0 = max3v(br, b, bl);
        float min0 = min3v(tl, t, tr);
        float max1 = max3v(tl, t, tr);
        float min1 = min3v(br, b, bl);
        float max2 = max3v(cc, l, b);
        float min2 = min3v(r, t, tr);
        float max3 = max3v(cc, r, t);
        float min3 = min3v(bl, l, b);
        float max4 = max3v(l, tl, bl);
        float min4 = min3v(r, br, tr);
        float max5 = max3v(r, br, tr);
        float min5 = min3v(l, tl, bl);
        float max6 = max3v(cc, l, t);
        float min6 = min3v(r, br, b);
        float max7 = max3v(cc, r, b);
        float min7 = min3v(t, l, tl);


        if (min0 > cc && min0 > max1) {
            get_largest(strength, image, lum, color,
                cc_ix, tl_ix, t_ix, tr_ix);
        } else if (min1 > cc && min1 > max1) {
            get_largest(strength, image, lum, color,
                cc_ix, br_ix, b_ix, bl_ix);
        }

        if (min2 > max2) {
            get_largest(strength, image, lum, color,
                cc_ix, r_ix, t_ix, tr_ix);
        } else if (min3 > max3) {
            get_largest(strength, image, lum, color,
                cc_ix, bl_ix, l_ix, b_ix);
        }

        if (min4 > cc && min4 > max4) {
            get_largest(strength, image, lum, color,
                cc_ix, r_ix, br_ix, tr_ix);
        } else if (min5 > cc && min5 > max5) {
            get_largest(strength, image, lum, color,
                cc_ix, l_ix, tl_ix, bl_ix);
        }

        if (min6 > max6) {
            get_largest(strength, image, lum, color,
                cc_ix, r_ix, br_ix, b_ix);
        } else if (min7 > max7) {
            get_largest(strength, image, lum, color,
                cc_ix, t_ix, l_ix, tl_ix);
        }


        preprocessed[3 * threadId] = color[0];
        preprocessed[3 * threadId + 1] = color[1];
        preprocessed[3 * threadId + 2] = color[2];
    }
}


__device__ __inline__ float 
clamp(float x, float lower, float upper)
{
    return min(upper,max(x,lower));
}

__device__ __inline__ void
compute_graident(float *lum, float *gradient, bool qualified)
{
    if (qualified) {
        int cc_ix = threadIdx.y*blockDim.x+threadIdx.x;
    
        /* [tl t tr]
         * [l cc  r]
         * [bl b br]
         */
        int t_ix = cc_ix - blockDim.x;
        int tl_ix = t_ix - 1;
        int tr_ix = t_ix + 1;
        int l_ix = cc_ix - 1;
        int r_ix = cc_ix + 1;
        int b_ix = cc_ix + blockDim.x;
        int bl_ix = b_ix - 1;
        int br_ix = b_ix + 1;
    
        float r = lum[r_ix];
        float l = lum[l_ix];
        float t = lum[t_ix];
        float tl = lum[tl_ix];
        float tr = lum[tr_ix];
        float b = lum[b_ix];
        float bl = lum[bl_ix];
        float br = lum[br_ix];
    
        /* Horizontal Gradient
        * [-1  0  1]
        * [-2  0  2]
        * [-1  0  1]
        */
        float xgrad = tr - tl + r + r - l - l + br - bl;
    
        /* Vertical Gradient
         * [-1 -2 -1]
         * [ 0  0  0]
         * [ 1  2  1]
         */
        float ygrad = bl - tl + b + b - t - t + br - tr;
    
        gradient[cc_ix] =
            1.0f - clamp(sqrt(xgrad * xgrad + ygrad * ygrad), 0.0f, 1.0f);
    }
}


__device__ __inline__ void 
get_average(float strength, float *image, float color[3],
    int cc, int a, int b, int c)
{   
    color[0] = image[cc * 3] * (1.0f - strength) +
        ((image[a * 3] + image[b * 3] + image[c * 3]) / 3.0f) * strength;
    color[1] = image[cc * 3 + 1] * (1.0f - strength) +
        ((image[a * 3 + 1] + image[b * 3 + 1] + image[c * 3 + 1]) / 3.0f) * strength;
    color[2] = image[cc * 3 + 2] * (1.0f - strength) +
        ((image[a * 3 + 2] + image[b * 3 + 2] + image[c * 3 + 2]) / 3.0f) * strength;
}


__device__ __inline__ void
push(float *image, float *gradient, float *pushed, bool qualified)
{
    int threadId = threadIdx.y*blockDim.x+threadIdx.x;
    float color[3];

    if (qualified) {
        color[0] = image[3 * threadId];
        color[1] = image[3 * threadId + 1];
        color[2] = image[3 * threadId + 2];

        int cc_ix = threadId;
        int r_ix = cc_ix + 1;
        int l_ix = cc_ix - 1;
        int t_ix = cc_ix - blockDim.x;
        int tl_ix = t_ix - 1;
        int tr_ix = t_ix + 1;
        int b_ix = cc_ix + blockDim.x;
        int bl_ix = b_ix - 1;
        int br_ix = b_ix + 1;

        float strength = cudaParam.strength_push;
        float cc = gradient[cc_ix];
        float r = gradient[r_ix];
        float l = gradient[l_ix];
        float t = gradient[t_ix];
        float tl = gradient[tl_ix];
        float tr = gradient[tr_ix];
        float b = gradient[b_ix];
        float bl = gradient[bl_ix];
        float br = gradient[br_ix];

        float max0 = max3v(br, b, bl);
        float min0 = min3v(tl, t, tr);
        float max1 = max3v(tl, t, tr);
        float min1 = min3v(br, b, bl);
        float max2 = max3v(cc, l, b);
        float min2 = min3v(r, t, tr);
        float max3 = max3v(cc, r, t);
        float min3 = min3v(bl, l, b);
        float max4 = max3v(l, tl, bl);
        float min4 = min3v(r, br, tr);
        float max5 = max3v(r, br, tr);
        float min5 = min3v(l, tl, bl);
        float max6 = max3v(cc, l, t);
        float min6 = min3v(r, br, b);
        float max7 = max3v(cc, r, b);
        float min7 = min3v(t, l, tl);

        if ( min0 > cc && min0 > max0) {
            get_average(strength, image, color,
                cc_ix, tl_ix, t_ix, tr_ix);
        } else if (min1 > cc &&  min1 > max1) {
            get_average(strength, image, color,
                cc_ix, br_ix, b_ix, bl_ix);
        } else if (min2 > max2) {
            get_average(strength, image, color,
                cc_ix, r_ix, t_ix, tr_ix);
        } else if (min3 > max3) {
            get_average(strength, image, color,
                cc_ix, bl_ix, l_ix, b_ix);
        } else if (min4 > cc && min4 > max4) {
            get_average(strength, image, color,
                cc_ix, r_ix, br_ix, tr_ix);
        } else if (min5 > cc && min5 > max5) {
            get_average(strength, image, color,
                cc_ix, l_ix, tl_ix, bl_ix);    
        } else if (min6 > max6) {
            get_average(strength, image, color,
                cc_ix, r_ix, br_ix, b_ix);
        } else if (min7 > max7) {
            get_average(strength, image, color,
                cc_ix, t_ix, l_ix, tl_ix);
        }

        pushed[3 * threadId] = color[0];
        pushed[3 * threadId + 1] = color[1];
        pushed[3 * threadId + 2] = color[2];
    }
}

__device__ __inline__ unsigned char
quantize(float x) 
{
    int r = x * 255;
    return min(255,max(r,0));
}

__device__ __inline__ void
output(float *image, unsigned char *dst, bool qualified)
{
    int threadId = threadIdx.y*blockDim.x+threadIdx.x;
    int pixelX = blockIdx.x*REGIONW+threadIdx.x-PADDING;
    int pixelY = blockIdx.y*REGIONH+threadIdx.y-PADDING;
    int dst_width = cudaParam.dst_width;
    int pixelId = pixelY*dst_width+pixelX;

    if (qualified) {
        dst[4*pixelId] = quantize(image[3*threadId]);
        dst[4*pixelId+1] = quantize(image[3*threadId+1]);
        dst[4*pixelId+2] = quantize(image[3*threadId+2]);
        dst[4*pixelId+3] = 255;
    
        // dst[4*pixelId] = quantize(image[threadId]);
        // dst[4*pixelId+1] = quantize(image[threadId]);
        // dst[4*pixelId+2] = quantize(image[threadId]);
        // dst[4*pixelId+3] = 255;
    }
}

__global__ void
kernel(unsigned char *src, unsigned char *dst)
{
    __shared__ float image[3*THREADW*THREADH];
    __shared__ float preprocessed[3*THREADW*THREADH];
    __shared__ float luminace[THREADW*THREADH];
    __shared__ float gradient[THREADW*THREADH];

    int pixelX = blockIdx.x*REGIONW+threadIdx.x-PADDING;
    int pixelY = blockIdx.y*REGIONH+threadIdx.y-PADDING;
    int dst_width = cudaParam.dst_width;
    int dst_height = cudaParam.dst_height;

    bool inside_image_nopaded = true;
    bool inside_image_padded = true;
    bool inside_thread_nopadded = true;
    bool inside_thread_onepadded = true;

    if (pixelX < 0 || pixelX >= dst_width ||
        pixelY < 0 || pixelY >= dst_height)
        inside_image_nopaded = false;

    if (pixelX >= dst_width+PADDING || pixelY >= dst_height+PADDING)
        inside_image_padded = false;

    if (threadIdx.x < PADDING || threadIdx.x >= blockDim.x - PADDING ||
        threadIdx.y < PADDING || threadIdx.y >= blockDim.y - PADDING) 
        inside_thread_nopadded = false;
    
    if (threadIdx.x < 1 || threadIdx.x >= blockDim.x - PADDING + 1 ||
        threadIdx.y < 1 || threadIdx.y >= blockDim.y - PADDING + 1) 
        inside_thread_onepadded = false;
    
    enlarge(src,image, inside_image_padded);
    compute_luminance(image, luminace, inside_image_padded);
    __syncthreads();
    preprocess(image,luminace,preprocessed, inside_image_padded && inside_thread_onepadded);
    compute_luminance(preprocessed, luminace, inside_image_padded);
    __syncthreads();
    compute_graident(luminace, gradient, inside_image_padded && inside_thread_onepadded);
    __syncthreads();
    push(preprocessed, gradient, image, inside_image_nopaded && inside_thread_nopadded);
    output(image, dst, inside_image_nopaded && inside_thread_nopadded);
}



